
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
	int deviceCount;
	int device;
	printf("Looking for CUDA devices...\n");
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
		printf("No devices found\n");
	}
	for(device = 0; device < deviceCount; ++device) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("device %d has compute capability %d.%d.\n",
			device, deviceProp.major, deviceProp.minor);
	}
}