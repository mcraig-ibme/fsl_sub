
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int device;
	for(device = 0; device < deviceCount; ++device) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("device %d has compute capability %d.%d.\n",
			device, deviceProp.major, deviceProp.minor);
	}
}